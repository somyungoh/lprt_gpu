#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include "GL/freeglut.h"	// -lGL -lglut

#define WIDTH   800
#define HEIGHT  600
#define tx      8
#define ty      8

// CUDA frame block
float *fb;
// GLUT display map
float *display_map;
 
// timing
float deltaTime;
float lastFrame;


using namespace std;


// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x*3 + i*3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}

// *************** GLUT Routine *********************


// void copy_to_display() {

//     for (int j = HEIGHT-1; j >= 0; j--) {
//         for (int i = 0; i < WIDTH; i++) {
//             size_t pixel_index = j*3*WIDTH + i*3;
            
//             float r = fb[pixel_index + 0];
//             float g = fb[pixel_index + 1];
//             float b = fb[pixel_index + 2];
//             int ir = int(255.99*r);
//             int ig = int(255.99*g);
//             int ib = int(255.99*b);
//         }
//     }
// }

// idle func
void idle() {

	glutPostRedisplay();
 	
	// compute time counts
	float currentFrame = 0.001f * glutGet(GLUT_ELAPSED_TIME); // since it is in milliseconds
	deltaTime = currentFrame - lastFrame;
	lastFrame = currentFrame;	
	
	// display current fps
	char title[32];
	float currentFPS = 1.f / deltaTime;
	sprintf(title, "LPRT - GPU \t %.5f fps", currentFPS);
	glutSetWindowTitle(title);
}

// keyboard input funtion
void keyboard(unsigned char key, int x, int y)
{
}


// arrowkey input
void arrowkey(int key, int x, int y)
{
	switch (key) {
	case GLUT_KEY_UP:
		//raytracer.cam_move(vec3(0, CAM_MOVE_ADD, 0));
		cout << "INPUT::camera orientation change" << endl;
		break;

	case GLUT_KEY_DOWN:
		//raytracer.cam_move(vec3(0, -1 * CAM_MOVE_ADD, 0));
		cout << "INPUT::camera orientation change" << endl;
		break;

	case GLUT_KEY_LEFT:
		//raytracer.cam_rot(-1 * CAM_ROT_ANGLE);
		//raytracer.cam_move(vec3(-1 * CAM_MOVE_ADD, 0, 0));
		cout << "INPUT::camera orientation change" << endl;
		break;

	case GLUT_KEY_RIGHT:
		//raytracer.cam_rot(CAM_ROT_ANGLE);
		//raytracer.cam_move(vec3(CAM_MOVE_ADD, 0, 0));
		cout << "INPUT::camera orientation change" << endl;
		break;

	default:
		break;
	}

	glutPostRedisplay();		// redraw
}


// resize function
void resize(int w, int h) {
	glViewport(0, 0, w, h);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0, (w / 2), 0, (h / 2), 0, 1);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
}


// display function
void display() {

	// raytracer.render();		// MAIN RENDERING ROUTINE
	// raytracer.get_image(display_map);

	// cout << "MAIN::Render finished." << endl;

    
    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(WIDTH/tx+1,HEIGHT/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, WIDTH, HEIGHT);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";


	glClear(GL_COLOR_BUFFER_BIT);
	glRasterPos2i(0, 0);
	glDrawPixels(WIDTH, HEIGHT, GL_RGB, GL_FLOAT, fb);
	glutSwapBuffers();

}


// **************************************************


int main(int argc, char* argv[]) {

    //
    // CUDA Routine
    //

    int nx = WIDTH;
    int ny = HEIGHT;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = 3*num_pixels*sizeof(float);

    // allocate FB
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // allocate display ap
    display_map = new float[WIDTH * HEIGHT * 3];

    
    //
    // GLUT Routine
    //

	//	GLUT Initialization
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
	glutInitWindowPosition(100, 100);
	glutInitWindowSize(WIDTH, HEIGHT);

	//	GLUT Display
	glutCreateWindow("LPRT_GPU");
	glutKeyboardFunc(keyboard);
	glutSpecialFunc(arrowkey);
	glutDisplayFunc(display);
	glutReshapeFunc(resize);
	glutIdleFunc(idle);

	glutMainLoop();


    checkCudaErrors(hipFree(fb));

    return 0;
}